#include "hip/hip_runtime.h"
#ifndef _SOMKERNELS_
#define _SOMKERNELS_

#include <math/ANFunctions.h>
#include <math/ANRandom.h>
#include <gpgpu/ANKernels.h>
#include <gpgpu/hip/hip_runtime_api.h>
#include <cfloat>

#include <cassert>
#include <cmath>

using namespace ANNGPGPU;


struct saxmy_functor {
	const float a;

	saxmy_functor(float _a) : a(_a) {}

	__host__ __device__
	float operator()(const float& x, const float& y) const { 
		return a * (x - y);
	}
};


// return the biggest of two tuples
struct bigger_tuple_functor {
    __device__ __host__
    thrust::tuple<float, unsigned int> operator() (	
    	const thrust::tuple<float, unsigned int> &a, 
		const thrust::tuple<float, unsigned int> &b ) 
    {
    	return (a >= b) ? a : b;
    }
};

// return the biggest of two tuples
struct smaller_tuple_functor {
    __device__ __host__
    thrust::tuple<float, unsigned int> operator() (	
    	const thrust::tuple<float, unsigned int> &a, 
		const thrust::tuple<float, unsigned int> &b ) 
    {
    	return (a <= b) ? a : b;
    }
};

float hostGetMax(const thrust::device_vector<float>& vec, unsigned int &ID) {
    // create implicit index sequence [0, 1, 2, ... ]
	thrust::counting_iterator<unsigned int> begin(0);
	thrust::counting_iterator<unsigned int> end(vec.size() );

    thrust::tuple<float, unsigned int> init(vec[0], 0);
    thrust::tuple<float, unsigned int> smallest;

    smallest = reduce( thrust::make_zip_iterator(make_tuple(vec.begin(), begin) ),
    				   thrust::make_zip_iterator(make_tuple(vec.end(), end) ),
                       init,
                       bigger_tuple_functor() );

    ID = thrust::get<1>(smallest);
    return vec[ID];
}

float hostGetMin(const thrust::device_vector<float>& vec, unsigned int &ID) {
    // create implicit index sequence [0, 1, 2, ... ]
	thrust::counting_iterator<unsigned int> begin(0);
	thrust::counting_iterator<unsigned int> end(vec.size() );
	
	thrust::tuple<float, unsigned int> init(vec[0], 0);
	thrust::tuple<float, unsigned int> smallest;
	
	smallest = reduce( thrust::make_zip_iterator(make_tuple(vec.begin(), begin) ),
					   thrust::make_zip_iterator(make_tuple(vec.end(), end) ),
					   init,
					   smaller_tuple_functor() );

	ID = thrust::get<1>(smallest);
    return vec[ID];
}
//////////////////////////////////////////////////////////////////////////////////////////////

struct minus_pow_functor {
    const float fVal;
    minus_pow_functor(float val) : fVal(val) {}

    __host__ __device__
	float operator()(const float& val) const { 
		return pow(fVal-val, 2);
	}
};

struct sqrt_functor {
    __host__ __device__
	float operator()(const float& val) const { 
		return sqrt(val);
	}
};
//////////////////////////////////////////////////////////////////////////////////////////////

struct bubble_functor {
	float fSigmaT;
	bubble_functor(const float &sigmaT) : fSigmaT(sigmaT)	{}

    __host__ __device__
	float operator()(const float& dist) const {
    	return ANN::fcn_bubble_neighborhood(dist, fSigmaT);
	}
};

struct gaussian_functor {
	float fSigmaT;
	gaussian_functor(const float &sigmaT) : fSigmaT(sigmaT)	{}

    __host__ __device__
	float operator()(const float& dist) const {
    	return ANN::fcn_gaussian_bell(dist, fSigmaT);
	}
};

struct cut_gaussian_functor {
	float fSigmaT;
	cut_gaussian_functor(const float &sigmaT) : fSigmaT(sigmaT)	{}

    __host__ __device__
	float operator()(const float& dist) const {
    	return ANN::fcn_cut_gaussian_bell(dist, fSigmaT);
	}
};

struct mexican_functor {
	float fSigmaT;
	mexican_functor(const float &sigmaT) : fSigmaT(sigmaT)	{}

    __host__ __device__
	float operator()(const float& dist) const {
    	return ANN::fcn_mexican_hat(dist, fSigmaT);
	}
};

struct epanechicov_functor {
	float fSigmaT;
	epanechicov_functor(const float &sigmaT) : fSigmaT(sigmaT)	{}

    __host__ __device__
	float operator()(const float& dist) const {
    	return ANN::fcn_epanechicov_neighborhood(dist, fSigmaT);
	}
};

struct hebbian_functor {
	float fLearningRate;
	float fInput;

	hebbian_functor(const float &learning_rate, const float &input) :
		fLearningRate(learning_rate), fInput(input) {}

    __host__ __device__
	float operator()(const float& fWeight, const float& fInfluence) const {
    	return fWeight + (fInfluence*fLearningRate*(fInput-fWeight) );
	}
};
//////////////////////////////////////////////////////////////////////////////////////////////

/*
 * Layout of SOMEdgeF2DArray:
 * 			COL1	COL2	COL3	COL(n+1)
 * ROW1		toNeur1	toNeur1	toNeur1	..
 * ROW2		toNeur2	toNeur2	toNeur2	..
 * ROW3		toNeur3	toNeur3	toNeur3	..
 * ROW(n+1)	..		..		..
 */
BMUExport
hostSOMFindBMNeuronID(std::vector<SplittedNetExport*> &SExp,
		const float &fConscienceRate)
{
	BMUExport retBMU;
	float fLastBMU = FLT_MAX;

	#pragma omp parallel for
	for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
		checkCudaErrors(hipSetDevice(iDev) );
		unsigned int BMUID = 0;

		unsigned int iWidth 	= SExp.at(iDev)->f2dEdges.GetW();
		unsigned int iHeight 	= SExp.at(iDev)->f2dEdges.GetH();

		assert(iWidth > 0);
		assert(iHeight > 0);

		thrust::device_vector<float> dvRes(iWidth, 0.f);
		thrust::device_vector<float> dvTmp(iWidth, 0.f);// temporary
		
		for(unsigned int y = 0; y < iHeight; y++) {
			thrust::transform(
				SExp.at(iDev)->f2dEdges.GetRowBegin(y),	// input
				SExp.at(iDev)->f2dEdges.GetRowEnd(y), 	// input
				dvTmp.begin(), 							// result
				minus_pow_functor((*SExp.at(iDev)->dvInput)[y]) ); 	// functor

			thrust::transform(
				dvRes.begin(), 						// input
				dvRes.end(), 						// input
				dvTmp.begin(),						// input
				dvRes.begin(), 						// result
				thrust::plus<float>() );			// functor
		}
		dvTmp = dvRes;

		// implementation of conscience mechanism
		if(fConscienceRate > 0.f) {
			thrust::device_vector<float> dvConscience(iWidth, 1.f / (float)iWidth);

			thrust::transform(
				dvConscience.begin(),
				dvConscience.end(),
				SExp.at(iDev)->dvConscience->begin(),
				dvConscience.begin(),
				thrust::minus<float>() );

			thrust::transform(
				dvRes.begin(),
				dvRes.end(),
				dvConscience.begin(),
				dvRes.begin(),
				thrust::minus<float>() );
		}

		thrust::transform(
			dvTmp.begin(),
			dvTmp.end(),
			SExp.at(iDev)->dvConscience->begin(),
			SExp.at(iDev)->dvConscience->begin(),
			saxmy_functor(fConscienceRate) );

		hostGetMin(dvRes, BMUID);

		// Check partial results for global BMU in all devices
		if(fLastBMU > dvRes[BMUID]) {
			fLastBMU = dvRes[BMUID];

			thrust::host_vector<float> vPos = SExp.at(iDev)->f2dPositions.GetSubArrayY(BMUID);
			retBMU = BMUExport(BMUID, iDev, vPos);
		}
	}
	
	return retBMU;
}

/*
 * Layout of SOMPositionF2DArray:
 * 			COL1	COL2	COL3	COL(n+1)
 * ROW1		Xpos	Xpos	Xpos	..
 * ROW2		Ypos	Ypos	Ypos	..
 * ROW3		Zpos	Zpos	Zpos	..
 * ROW(n+1)	..		..		..		..
 */
template<typename BinaryFunction>
void hostSOMPropagateBW( std::vector<SplittedNetExport*> &SExp,
		const BMUExport &BMU,
		const float &fSigmaT,
		const float &fLearningRate,
		const BinaryFunction &binaryDistFunc
		)
{
	#pragma omp parallel for
	for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
		checkCudaErrors(hipSetDevice(iDev) );
		
		unsigned int iWidth 	= SExp.at(iDev)->f2dPositions.GetW();
		unsigned int iHeight 	= SExp.at(iDev)->f2dPositions.GetH();

		thrust::device_vector<float> dvBMUPos = BMU.dvBMUPos;
		thrust::device_vector<float> dvTmp(iWidth, 0.f); // temporary
		thrust::device_vector<float> dvInfluence(iWidth, 0.f);
		thrust::device_vector<float> dvDist(iWidth, 0.f);

		// 1. Calc distances for all neurons to BMNeuron
		// Distance = sqrt(pow(x,2)+pow(y,2)+pow(z,2)+pow(n+1,2) );
		for(unsigned int y = 0; y < iHeight; y++) { 	// for each coordinate position of the neuron
			thrust::transform(
				SExp.at(iDev)->f2dPositions.GetRowBegin(y),		// input
				SExp.at(iDev)->f2dPositions.GetRowEnd(y), 		// input
				dvTmp.begin(), 						// result
				minus_pow_functor(dvBMUPos[y]) ); 			// functor

			thrust::transform(
				dvDist.begin(), 					// input
				dvDist.end(), 						// input
				dvTmp.begin(),						// input
				dvDist.begin(), 					// result
				thrust::plus<float>() );				// functor
		}
		thrust::transform(
			dvDist.begin(),							// input
			dvDist.end(), 							// input
			dvDist.begin(), 						// result
			sqrt_functor() );						// functor

		// 2. Calculate the influence for each neuron
		thrust::transform(
			dvDist.begin(),							// input
			dvDist.end(), 							// input
			dvInfluence.begin(), 						// result
			binaryDistFunc );					// functor

		// 3. Only handle neurons in radius:
		// 3a. Make stencil
		dvTmp.assign(iWidth, fSigmaT);
		thrust::transform(
			dvDist.begin(), 						// input 1
			dvDist.end(),							// input 1
			dvTmp.begin(),							// input 1
			dvTmp.begin(), 							// result
			thrust::less<float>() 						// functor
		);

		// 3b. Use stencil to modify only neurons inside the radius
		// Save result in the ANN::F2DArray
		iWidth 	= SExp.at(iDev)->f2dEdges.GetW();
		iHeight = SExp.at(iDev)->f2dEdges.GetH();

		for(unsigned int y = 0; y < iHeight; y++) {			// for each edge of the neuron
			thrust::transform_if(
				SExp.at(iDev)->f2dEdges.GetRowBegin(y),		// input 1
				SExp.at(iDev)->f2dEdges.GetRowEnd(y), 		// input 1
				dvInfluence.begin(),						// input 2
				dvTmp.begin(),								// stencil
				SExp.at(iDev)->f2dEdges.GetRowBegin(y), 		// result
				hebbian_functor(fLearningRate, (*SExp.at(iDev)->dvInput)[y]), // functor
				thrust::identity<int>() ); 					// predicate
		}
	}
}

void hostSOMTraining( std::vector<SplittedNetExport*> &SExp,
		const ANN::TrainingSet &InputSet,
		const unsigned int &iCycles,
		const float &fSigma0, 
		const float &fLearningRate0,
		const float &fConscienceRate,
		float (*pfnDecay)(const float &, const float &, const float &),
		const ANN::DistFunction &DistFunc )
{
	float fLambda 	= iCycles / log(fSigma0);

	int iMin 		= 0;
	int iMax 		= InputSet.GetNrElements()-1;
	unsigned int iProgCount = 1;

	// use 8 proximal neurons as standard
	float fSigmaT = sqrt(2.f);

	for(unsigned int i = 0; i < iCycles; i++) {
		if(iCycles >= 10) {
			if(((i+1) / (iCycles/10)) == iProgCount && (i+1) % (iCycles/10) == 0) {
				std::cout<<"Current training progress calculated by the GPU is: "<<iProgCount*10.f<<"%/Step="<<i+1<<std::endl;
				iProgCount++;
			}
		}
		else {
			std::cout<<"Current training progress calculated by the CPU is: "<<(float)(i+1.f)/(float)iCycles*100.f<<"%/Step="<<i+1<<std::endl;
		}

		// Set input
		std::vector<float> vCurInput = InputSet.GetInput(ANN::RandInt(iMin, iMax) );
		
		#pragma omp parallel for
		for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
			checkCudaErrors(hipSetDevice(iDev) );
			thrust::device_vector<float> *p_dvInputVector = new thrust::device_vector<float>(vCurInput.size() );
			thrust::copy(vCurInput.begin(), vCurInput.end(), p_dvInputVector->begin() );
			SExp[iDev]->dvInput = p_dvInputVector;
		}

		// Find BMNeuron
		BMUExport BMUExp = hostSOMFindBMNeuronID(SExp, fConscienceRate);

		// Calc m_fSigmaT if conscience is _not_ used
		if(fConscienceRate <= 0.f) {
			fSigmaT = std::floor(pfnDecay(fSigma0, i, fLambda) + 0.5f);
		}
		float fLearningRate = pfnDecay(fLearningRate0, i, iCycles);

		// Propagate BW
		if (strcmp (DistFunc.name, "gaussian") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					gaussian_functor(fSigmaT)); 	// const
		}
		else if (strcmp (DistFunc.name, "mexican") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					mexican_functor(fSigmaT)); 	// const
		}
		else if (strcmp (DistFunc.name, "bubble") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					bubble_functor(fSigmaT)); 	// const
		}
		else if (strcmp (DistFunc.name, "cut_gaussian") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					cut_gaussian_functor(fSigmaT)); // const
		}
		else if (strcmp (DistFunc.name, "epanechicov") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					epanechicov_functor(fSigmaT)); 	// const
		}
	}
}

#endif
