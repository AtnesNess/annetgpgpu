#include "hip/hip_runtime.h"
#ifndef _SOMKERNELS_
#define _SOMKERNELS_

#include <math/ANFunctions.h>
#include <math/ANRandom.h>
#include <gpgpu/ANKernels.h>

#include <cassert>
#include <cmath>


struct saxmy_functor {
	const float a;

	saxmy_functor(float _a) : a(_a) {}

	__host__ __device__
	float operator()(const float& x, const float& y) const { 
		return a * (x - y);
	}
};


// return the biggest of two tuples
struct bigger_tuple_functor {
    __device__ __host__
    thrust::tuple<float, unsigned int> operator() (	
    	const thrust::tuple<float, unsigned int> &a, 
		const thrust::tuple<float, unsigned int> &b ) 
    {
    	return (a >= b) ? a : b;
    }
};

// return the biggest of two tuples
struct smaller_tuple_functor {
    __device__ __host__
    thrust::tuple<float, unsigned int> operator() (	
    	const thrust::tuple<float, unsigned int> &a, 
		const thrust::tuple<float, unsigned int> &b ) 
    {
    	return (a <= b) ? a : b;
    }
};

float hostGetMax(const thrust::device_vector<float>& vec, unsigned int &ID) {
    // create implicit index sequence [0, 1, 2, ... ]
	thrust::counting_iterator<unsigned int> begin(0);
	thrust::counting_iterator<unsigned int> end(vec.size() );

    thrust::tuple<float, unsigned int> init(vec[0], 0);
    thrust::tuple<float, unsigned int> smallest;

    smallest = reduce( thrust::make_zip_iterator(make_tuple(vec.begin(), begin) ),
    				   thrust::make_zip_iterator(make_tuple(vec.end(), end) ),
                       init,
                       bigger_tuple_functor() );

    ID = thrust::get<1>(smallest);
    return vec[ID];
}

float hostGetMin(const thrust::device_vector<float>& vec, unsigned int &ID) {
    // create implicit index sequence [0, 1, 2, ... ]
	thrust::counting_iterator<unsigned int> begin(0);
	thrust::counting_iterator<unsigned int> end(vec.size() );
	
	thrust::tuple<float, unsigned int> init(vec[0], 0);
	thrust::tuple<float, unsigned int> smallest;
	
	smallest = reduce( thrust::make_zip_iterator(make_tuple(vec.begin(), begin) ),
					   thrust::make_zip_iterator(make_tuple(vec.end(), end) ),
					   init,
					   smaller_tuple_functor() );

	ID = thrust::get<1>(smallest);
    return vec[ID];
}
//////////////////////////////////////////////////////////////////////////////////////////////

/**
 * 
 */
struct minus_pow_functor {
    const float fVal;
    minus_pow_functor(float val) : fVal(val) {}

    __host__ __device__
	float operator()(const float& val) const { 
		return pow(fVal-val, 2);
	}
};

struct sqrt_functor {
    __host__ __device__
	float operator()(const float& val) const { 
		return sqrt(val);
	}
};

/*
 * Layout of SOMEdgeMatrix:
 * 			COL1	COL2	COL3	COL(n+1)
 * ROW1		toNeur1	toNeur1	toNeur1	..
 * ROW2		toNeur2	toNeur2	toNeur2	..
 * ROW3		toNeur3	toNeur3	toNeur3	..
 * ROW(n+1)	..		..		..
 */
unsigned int hostSOMFindBMNeuronID( thrust::device_vector<float> &ConscienceVector,
		const ANN::Matrix &SOMEdgeMatrix, 
		const thrust::device_vector<float> &InputVector,
		const float &fConscienceRate) 
{
	unsigned int BMUID 		= 0;
	unsigned int iWidth 	= SOMEdgeMatrix.getW();
	unsigned int iHeight 	= SOMEdgeMatrix.getH();
	
	assert(iWidth > 0);
	assert(iHeight > 0);
	
	thrust::device_vector<float> dvRes(iWidth, 0.f);
	thrust::device_vector<float> dvConscience(iWidth, -1.f / (float)iWidth);
	thrust::device_vector<float> dvTmp(iWidth, 0.f); // temporary
	
	for(unsigned int y = 0; y < iHeight; y++) {
		thrust::transform(
				SOMEdgeMatrix.getRowBegin(y),		// input
				SOMEdgeMatrix.getRowEnd(y), 		// input
				dvTmp.begin(), 						// result
				minus_pow_functor(InputVector[y]) ); // functor

		thrust::transform(
				dvRes.begin(), 						// input
				dvRes.end(), 						// input
				dvTmp.begin(),						// input
				dvRes.begin(), 						// result
				thrust::plus<float>() );			// functor
	}

	// implementation of conscience mechanism
	dvTmp = dvRes;
	if(fConscienceRate > 0.f) {
		thrust::transform(
			ConscienceVector.begin(), 
			ConscienceVector.end(), 
			dvConscience.begin(), 
			dvConscience.begin(), 
			thrust::plus<float>() );

		thrust::transform(
				dvConscience.begin(),
				dvConscience.end(),
				dvRes.begin(),
				dvRes.begin(),
				thrust::plus<float>() );
	}

	thrust::transform(
		dvTmp.begin(),
		dvTmp.end(),
		ConscienceVector.begin(),
		ConscienceVector.begin(),
		saxmy_functor(fConscienceRate) );

/*
	thrust::transform(
			dvRes.begin(),							// input
			dvRes.end(), 							// input
			dvRes.begin(), 							// result
			sqrt_functor() );						// functor
*/
	hostGetMin(dvRes, BMUID);
	
//	dvRes.clear();									// cleanup
//	dvTmp.clear();									// cleanup
	
	return BMUID;
}

/* // TODO fucking need better GRAKA
struct dist_functor {
	float fSigmaT;
	float (*pf_distance)(const float &, const float &);
	dist_functor(float (*distance)(const float &, const float &), const float &sigmaT) : pf_distance(distance), fSigmaT(sigmaT)	{}
	
    __host__ __device__
	float operator()(const float& val) const { 
		return (pf_distance)(val, fSigmaT);
	}
};
*/ // TODO fucking need better GRAKA

struct gaussian_bell_functor {
	float fSigmaT;
	gaussian_bell_functor(const float &sigmaT) : fSigmaT(sigmaT)	{}
	
    __host__ __device__
	float operator()(const float& dist) const { 
    	return ANN::fcn_gaussian_bell(dist, fSigmaT);
	}
};

struct hebbian_functor {
	float fLearningRate;
	float fInput;
	
	hebbian_functor(const float &learning_rate, const float &input) : 
		fLearningRate(learning_rate), fInput(input) {}
	
    __host__ __device__
	float operator()(const float& fWeight, const float& fInfluence) const { 
    	return fWeight + (fInfluence*fLearningRate*(fInput-fWeight) );
	}
};

/*
 * Layout of SOMPositionMatrix:
 * 			COL1	COL2	COL3	COL(n+1)
 * ROW1		Xpos	Xpos	Xpos	..
 * ROW2		Ypos	Ypos	Ypos	..
 * ROW3		Zpos	Zpos	Zpos	..
 * ROW(n+1)	..		..		..		..
 */
void hostSOMPropagateBW( ANN::Matrix &SOMEdgeMatrix,
		const ANN::Matrix &SOMPositionMatrix, 
		const thrust::device_vector<float> &dvInputVector,
		const unsigned int BMUID, 
		const float &fSigmaT, 
		const float &fLearningRate
		) 
{
	unsigned int iWidth 	= SOMPositionMatrix.getW();
	unsigned int iHeight 	= SOMPositionMatrix.getH();
	
	// TODO PUT this in the TRAINING function the increase performance
	thrust::device_vector<float> dvBMUPos = SOMPositionMatrix.getCol(BMUID);
	thrust::device_vector<float> dvTmp(iWidth, 0.f); // temporary
	thrust::device_vector<float> dvInfluence(iWidth, 0.f); 
	thrust::device_vector<float> dvDist(iWidth, 0.f);
	
	// 1. Calc distances for all neurons to BMNeuron
	// Distance = sqrt(pow(x,2)+pow(y,2)+pow(z,2)+pow(n+1,2) );
	for(unsigned int y = 0; y < iHeight; y++) { 	// for each coordinate position of the neuron
		thrust::transform(
				SOMPositionMatrix.getRowBegin(y),	// input
				SOMPositionMatrix.getRowEnd(y), 	// input
				dvTmp.begin(), 						// result
				minus_pow_functor(dvBMUPos[y]) ); 	// functor
		
		thrust::transform(
				dvDist.begin(), 					// input
				dvDist.end(), 						// input
				dvTmp.begin(),						// input
				dvDist.begin(), 					// result
				thrust::plus<float>() );			// functor
	}
	thrust::transform(
			dvDist.begin(),							// input
			dvDist.end(), 							// input
			dvDist.begin(), 						// result
			sqrt_functor() );						// functor
	
	// 2. Calculate the influence for each neuron
	thrust::transform(
			dvDist.begin(),							// input
			dvDist.end(), 							// input
			dvInfluence.begin(), 					// result
			gaussian_bell_functor(fSigmaT) );		// functor
	
	// 3. Only handle neurons in radius:
	// 3a. Make stencil
	dvTmp.assign(iWidth, fSigmaT);
	thrust::transform(
			dvDist.begin(), 						// input 1
			dvDist.end(),							// input 1
			dvTmp.begin(),							// input 1
			dvTmp.begin(), 							// result
			thrust::less_equal<float>() 			// functor
	);
	
	// 3b. Use stencil to modify only neurons inside the radius
	// Save result in the ANN::Matrix
	iWidth 	= SOMEdgeMatrix.getW();
	iHeight = SOMEdgeMatrix.getH();

	for(unsigned int y = 0; y < iHeight; y++) {		// for each edge of the neuron   	
		thrust::transform_if(
				SOMEdgeMatrix.getRowBegin(y),		// input 1
				SOMEdgeMatrix.getRowEnd(y), 		// input 1
				dvInfluence.begin(),				// input 2
				dvTmp.begin(),						// stencil
				SOMEdgeMatrix.getRowBegin(y), 		// result
				hebbian_functor(fLearningRate, dvInputVector[y]), // functor
				thrust::identity<int>() ); 			// predicate
	}
	
	// 4. Clean!
//	dvBMUPos.clear();
//	dvTmp.clear(); 									// cleanup
//	dvInfluence.clear(); 							// cleanup
//	dvDist.clear(); 								// cleanup
}

void hostSOMTraining( thrust::device_vector<float> &ConscienceVector,
		ANN::Matrix &SOMEdgeMatrix,
		const ANN::Matrix &SOMPositionMatrix, 
		const ANN::TrainingSet &InputSet,
		const unsigned int &iCycles,
		const float &fSigma0, 
		const float &fLearningRate0,
		const float &fConscienceRate,
		float (*pfnDecay)(const float &, const float &, const float &) )
{
	float fLambda 	= iCycles / log(fSigma0);
	
	int iMin 		= 0;
	int iMax 		= InputSet.GetNrElements()-1;
	unsigned int iProgCount = 1;
	
	for(unsigned int i = 0; i < iCycles; i++) {
		if(iCycles >= 10) {
			if(((i+1) / (iCycles/10)) == iProgCount && (i+1) % (iCycles/10) == 0) {
				std::cout<<"Current training progress calculated by the GPU is: "<<iProgCount*10.f<<"%/Step="<<i+1<<std::endl;
				iProgCount++;
			}
		}
		else {
			std::cout<<"Current training progress calculated by the CPU is: "<<(float)(i+1.f)/(float)iCycles*100.f<<"%/Step="<<i+1<<std::endl;
		}
		// Set input
		std::vector<float> vCurInput = InputSet.GetInput(ANN::RandInt(iMin, iMax) );
		thrust::device_vector<float> dvInputVector(vCurInput.size() );
		thrust::copy(vCurInput.begin(), vCurInput.end(), dvInputVector.begin() );
		
		// Find BMNeuron
		unsigned int BMUID = hostSOMFindBMNeuronID(ConscienceVector, SOMEdgeMatrix, dvInputVector, fConscienceRate);

		// use 8 proximal neurons as standard 
		float fSigmaT = sqrt(2.f);
		// Calc m_fSigmaT if conscience is _not_ used
		if(fConscienceRate == 0.f)
			fSigmaT = pfnDecay(fSigma0, i, fLambda);
		float fLearningRate = pfnDecay(fLearningRate0, i, iCycles);
		
		// Propagate BW
		hostSOMPropagateBW( SOMEdgeMatrix,
				SOMPositionMatrix, 	// const
				dvInputVector,		// const
				BMUID,			// const
				fSigmaT,		// const
				fLearningRate ); 	// const
	}
}

#endif
