#include "hip/hip_runtime.h"
#ifndef _SOMKERNELS_
#define _SOMKERNELS_

#include "include/math/Random.h"
#include "include/gpgpu/Kernels.h"
#include "include/gpgpu/Functors.h"
#include "include/gpgpu/hip/hip_runtime_api.h"

#include <cfloat>
#include <cassert>
#include <cmath>

//#include <omp.h>

using namespace ANNGPGPU;


float hostGetMax(const thrust::device_vector<float>& vec, unsigned int &ID) {
	// create implicit index sequence [0, 1, 2, ... ]
	thrust::counting_iterator<unsigned int> begin(0);
	thrust::counting_iterator<unsigned int> end(vec.size() );
	thrust::tuple<float, unsigned int> init(vec[0], 0);
	thrust::tuple<float, unsigned int> smallest;

	smallest = reduce( thrust::make_zip_iterator(make_tuple(vec.begin(), begin) ),
			thrust::make_zip_iterator(make_tuple(vec.end(), end) ),
			init,
			bigger_tuple_functor() );

	ID = thrust::get<1>(smallest);
	return vec[ID];
}

float hostGetMin(const thrust::device_vector<float>& vec, unsigned int &ID) {
	// create implicit index sequence [0, 1, 2, ... ]
	thrust::counting_iterator<unsigned int> begin(0);
	thrust::counting_iterator<unsigned int> end(vec.size() );
	thrust::tuple<float, unsigned int> init(vec[0], 0);
	thrust::tuple<float, unsigned int> smallest;
	
	smallest = reduce( thrust::make_zip_iterator(make_tuple(vec.begin(), begin) ),
			thrust::make_zip_iterator(make_tuple(vec.end(), end) ),
			init,
			smaller_tuple_functor() );

	ID = thrust::get<1>(smallest);
	return vec[ID];
}
//////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////
/*
 * Layout of SOMEdgeF2DArray:
 * 		COL1	COL2	COL3	COL(n+1)
 * ROW1		toNeur1	toNeur1	toNeur1	..
 * ROW2		toNeur2	toNeur2	toNeur2	..
 * ROW3		toNeur3	toNeur3	toNeur3	..
 * ROW(n+1)	..		..		..
 */
BMUExport
hostSOMFindBMNeuronID(std::vector<SplittedNetExport*> &SExp,
		const float &fConscienceRate)
{
	BMUExport retBMU;
	float fLastBMU = FLT_MAX;

	omp_set_num_threads(SExp.size() );  	// create as many CPU threads as there are CUDA devices
	#pragma omp parallel 			//for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
	{
		unsigned int iDev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(iDev) );
		unsigned int BMUID = 0;

		unsigned int iWidth 	= SExp.at(iDev)->f2dEdges.GetW();
		unsigned int iHeight 	= SExp.at(iDev)->f2dEdges.GetH();

		assert(iWidth 	> 0);
		assert(iHeight 	> 0);

		thrust::device_vector<float> dvRes(iWidth, 0.f);
		thrust::device_vector<float> dvTmp(iWidth, 0.f); 			// temporary
		thrust::device_vector<float> dvConscience(iWidth, 1.f / (float)iWidth);

		for(unsigned int y = 0; y < iHeight; y++) {
			thrust::transform(
				SExp.at(iDev)->f2dEdges.GetRowBegin(y),			// input
				SExp.at(iDev)->f2dEdges.GetRowEnd(y), 			// input
				dvTmp.begin(), 						// result
				minus_pow_functor((*SExp.at(iDev)->dvInput)[y]) ); 	// functor

			thrust::transform(
				dvRes.begin(), 						// input
				dvRes.end(), 						// input
				dvTmp.begin(),						// input
				dvRes.begin(), 						// result
				thrust::plus<float>() );				// functor
		}

		// implementation of conscience mechanism
		if(fConscienceRate > 0.f) {
			thrust::transform(
				dvConscience.begin(),
				dvConscience.end(),
				SExp.at(iDev)->dvConscience->begin(),
				dvConscience.begin(),
				thrust::minus<float>() );

			thrust::transform(
				dvRes.begin(),
				dvRes.end(),
				dvConscience.begin(),
				dvRes.begin(),
				thrust::minus<float>() );
		}

		thrust::transform(
			dvRes.begin(),
			dvRes.end(),
			SExp.at(iDev)->dvConscience->begin(),
			SExp.at(iDev)->dvConscience->begin(),
			saxmy_functor(fConscienceRate) );

		hostGetMin(dvRes, BMUID);

		// Check partial results for global BMU in all devices
		if(fLastBMU > dvRes[BMUID]) {
			fLastBMU = dvRes[BMUID];
			thrust::host_vector<float> vPos = SExp.at(iDev)->f2dPositions.GetSubArrayY(BMUID);
			retBMU = BMUExport(BMUID, iDev, vPos);
		}
	}
	return retBMU;
}

/*
 * Layout of SOMPositionF2DArray:
 * 		COL1	COL2	COL3	COL(n+1)
 * ROW1		Xpos	Xpos	Xpos	..
 * ROW2		Ypos	Ypos	Ypos	..
 * ROW3		Zpos	Zpos	Zpos	..
 * ROW(n+1)	..		..		..		..
 */
template<typename BinaryFunction>
void hostSOMPropagateBW( std::vector<SplittedNetExport*> &SExp,
		const BMUExport &BMU,
		const float &fSigmaT,
		const float &fLearningRate,
		const BinaryFunction &binaryDistFunc
		)
{
	omp_set_num_threads(SExp.size() );  	// create as many CPU threads as there are CUDA devices
	#pragma omp parallel 			//for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
	{
		unsigned int iDev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(iDev) );
		
		unsigned int iWidth 	= SExp.at(iDev)->f2dPositions.GetW();
		unsigned int iHeight 	= SExp.at(iDev)->f2dPositions.GetH();

		thrust::device_vector<float> dvTmp(iWidth, 0.f); 			// temporary
		thrust::device_vector<float> dvInfluence(iWidth, 0.f);
		thrust::device_vector<float> dvDist(iWidth, 0.f);

		// 1. Calc distances for all neurons to BMNeuron
		// Distance = sqrt(pow(x,2)+pow(y,2)+pow(z,2)+pow(n+1,2) );
		for(int y = 0; y < static_cast<int>(iHeight); y++) { 				// for each coordinate position of the neuron
			thrust::transform(
				SExp.at(iDev)->f2dPositions.GetRowBegin(y),		// input
				SExp.at(iDev)->f2dPositions.GetRowEnd(y), 		// input
				dvTmp.begin(), 						// result
				minus_pow_functor(BMU.dvBMUPos[y]) ); 			// functor

			thrust::transform(
				dvDist.begin(), 					// input
				dvDist.end(), 						// input
				dvTmp.begin(),						// input
				dvDist.begin(), 					// result
				thrust::plus<float>() );				// functor
		}

		thrust::transform(
			dvDist.begin(),							// input
			dvDist.end(), 							// input
			dvDist.begin(), 						// result
			sqrt_functor() );						// functor

		// 2. Calculate the influence for each neuron
		thrust::transform(
			dvDist.begin(),							// input
			dvDist.end(), 							// input
			dvInfluence.begin(), 						// result
			binaryDistFunc );						// functor

		// 3. Only handle neurons in radius:
		// 3a. Make stencil
		dvTmp.assign(iWidth, fSigmaT);
		thrust::transform(
			dvDist.begin(), 						// input 1
			dvDist.end(),							// input 1
			dvTmp.begin(),							// input 1
			dvTmp.begin(), 							// result
			thrust::less<float>() 						// functor
		);

		// 3b. Use stencil to modify only neurons inside the radius
		// Save result in the ANN::F2DArray
		iWidth 	= SExp.at(iDev)->f2dEdges.GetW();
		iHeight = SExp.at(iDev)->f2dEdges.GetH();

		for(int y = 0; y < static_cast<int>(iHeight); y++) {				// for each edge of the neuron
			thrust::transform_if(
				SExp.at(iDev)->f2dEdges.GetRowBegin(y),			// input 1
				SExp.at(iDev)->f2dEdges.GetRowEnd(y), 			// input 1
				dvInfluence.begin(),					// input 2
				dvTmp.begin(),						// stencil
				SExp.at(iDev)->f2dEdges.GetRowBegin(y), 		// result
				hebbian_functor(fLearningRate, (*SExp.at(iDev)->dvInput)[y]), // functor
				thrust::identity<int>() ); 				// predicate
		}
	}
}

void hostSOMTraining( std::vector<SplittedNetExport*> &SExp,
		const ANN::TrainingSet &InputSet,
		const unsigned int &iCycles,
		const float &fSigma0, 
		const float &fLearningRate0,
		const float &fConscienceRate,
		float (*pfnDecay)(const float &, const float &, const float &),
		const ANN::DistFunction &DistFunc )
{
	float fLambda 	= iCycles / log(fSigma0);

	int iMin 		= 0;
	int iMax 		= InputSet.GetNrElements()-1;
	unsigned int iProgCount = 1;

	// use 8 proximal neurons as standard
	float fSigmaT = sqrt(2.f);

	for(unsigned int i = 0; i < iCycles; i++) {
		if(iCycles >= 10) {
			if(((i+1) / (iCycles/10)) == iProgCount && (i+1) % (iCycles/10) == 0) {
				std::cout<<"Current training progress calculated by the GPU is: "<<iProgCount*10.f<<"%/Step="<<i+1<<std::endl;
				iProgCount++;
			}
		}
		else {
			std::cout<<"Current training progress calculated by the CPU is: "<<(float)(i+1.f)/(float)iCycles*100.f<<"%/Step="<<i+1<<std::endl;
		}

		// Set input
		std::vector<float> vCurInput = InputSet.GetInput(ANN::RandInt(iMin, iMax) );
		
		for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
			checkCudaErrors(hipSetDevice(iDev) );
			thrust::device_vector<float> *p_dvInputVector = new thrust::device_vector<float>(vCurInput.size() );
			thrust::copy(vCurInput.begin(), vCurInput.end(), p_dvInputVector->begin() );
			SExp[iDev]->dvInput = p_dvInputVector;
		}

		// Find BMNeuron
		BMUExport BMUExp = hostSOMFindBMNeuronID(SExp, fConscienceRate);

		// Calc m_fSigmaT if conscience is _not_ used
		if(fConscienceRate <= 0.f) {
			fSigmaT = std::floor(pfnDecay(fSigma0, i, fLambda) + 0.5f);
		}
		float fLearningRate = pfnDecay(fLearningRate0, i, iCycles);

		// Propagate BW
		if (strcmp (DistFunc.name, "gaussian") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					gaussian_functor(fSigmaT)); 	// const
		}
		else if (strcmp (DistFunc.name, "mexican") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					mexican_functor(fSigmaT)); 	// const
		}
		else if (strcmp (DistFunc.name, "bubble") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					bubble_functor(fSigmaT)); 	// const
		}
		else if (strcmp (DistFunc.name, "cut_gaussian") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					cut_gaussian_functor(fSigmaT)); // const
		}
		else if (strcmp (DistFunc.name, "epanechicov") == 0) {
			hostSOMPropagateBW( SExp,
					BMUExp,				// const
					fSigmaT,			// const
					fLearningRate,
					epanechicov_functor(fSigmaT)); 	// const
		}
	}
}

#endif
